// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *x, int incx, int *res) {
  // Start
  hipblasIsamin(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
               incx /*int*/, res /*int **/);
  // End
}
