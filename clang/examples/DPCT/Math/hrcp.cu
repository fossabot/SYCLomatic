// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h, __hip_bfloat16 b) {
  // Start
  hrcp(h /*__half*/);
  hrcp(b /*__nv_bfloat16*/);
  // End
}
