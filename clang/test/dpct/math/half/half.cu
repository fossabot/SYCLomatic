// RUN: dpct --format-range=none -out-root %T/math/half/half %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/math/half/half/half.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/math/half/half/half.dp.cpp -o %T/math/half/half/half.dp.o %}


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

__global__ void kernelFuncHalfConversion() {
  float f;
  float2 f2;
  half h;
  half2 h2;
  int i;
  long long ll;
  short s;
  unsigned u;
  unsigned long long ull;
  unsigned short us;
  // CHECK: h2 = f2.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __float22half2_rn(f2);
  // CHECK: h = sycl::vec<float, 1>(f).convert<sycl::half, sycl::rounding_mode::automatic>()[0];
  h = __float2half(f);
  // CHECK: h2 = sycl::float2(f).convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __float2half2_rn(f);
  // CHECK: h = sycl::vec<float, 1>(f).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __float2half_rd(f);
  // sycl::vec<float, 1>(f).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  __float2half_rn(f);
  // CHECK: h = sycl::vec<float, 1>(f).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __float2half_ru(f);
  // CHECK: h = sycl::vec<float, 1>(f).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __float2half_rz(f);
  // CHECK: h2 = sycl::float2(f, f).convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __floats2half2_rn(f, f);
  // CHECK: f2 = h2.convert<float, sycl::rounding_mode::automatic>();
  f2 = __half22float2(h2);
  // CHECK: f = sycl::vec<sycl::half, 1>(h).convert<float, sycl::rounding_mode::automatic>()[0];
  f = __half2float(h);
  // CHECK: h2 = sycl::half2(h);
  h2 = __half2half2(h);
  // CHECK: i = sycl::vec<sycl::half, 1>(h).convert<int, sycl::rounding_mode::rtn>()[0];
  i = __half2int_rd(h);
  // CHECK: i = sycl::vec<sycl::half, 1>(h).convert<int, sycl::rounding_mode::rte>()[0];
  i = __half2int_rn(h);
  // CHECK: i = sycl::vec<sycl::half, 1>(h).convert<int, sycl::rounding_mode::rtp>()[0];
  i = __half2int_ru(h);
  // CHECK: i = sycl::vec<sycl::half, 1>(h).convert<int, sycl::rounding_mode::rtz>()[0];
  i = __half2int_rz(h);
  // CHECK: ll = sycl::vec<sycl::half, 1>(h).convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __half2ll_rd(h);
  // CHECK: ll = sycl::vec<sycl::half, 1>(h).convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __half2ll_rn(h);
  // CHECK: ll = sycl::vec<sycl::half, 1>(h).convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __half2ll_ru(h);
  // CHECK: ll = sycl::vec<sycl::half, 1>(h).convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __half2ll_rz(h);
  // CHECK: s = sycl::vec<sycl::half, 1>(h).convert<short, sycl::rounding_mode::rtn>()[0];
  s = __half2short_rd(h);
  // CHECK: s = sycl::vec<sycl::half, 1>(h).convert<short, sycl::rounding_mode::rte>()[0];
  s = __half2short_rn(h);
  // CHECK: s = sycl::vec<sycl::half, 1>(h).convert<short, sycl::rounding_mode::rtp>()[0];
  s = __half2short_ru(h);
  // CHECK: s = sycl::vec<sycl::half, 1>(h).convert<short, sycl::rounding_mode::rtz>()[0];
  s = __half2short_rz(h);
  // CHECK: u = sycl::vec<sycl::half, 1>(h).convert<unsigned, sycl::rounding_mode::rtn>()[0];
  u = __half2uint_rd(h);
  // CHECK: u = sycl::vec<sycl::half, 1>(h).convert<unsigned, sycl::rounding_mode::rte>()[0];
  u = __half2uint_rn(h);
  // CHECK: u = sycl::vec<sycl::half, 1>(h).convert<unsigned, sycl::rounding_mode::rtp>()[0];
  u = __half2uint_ru(h);
  // CHECK: u = sycl::vec<sycl::half, 1>(h).convert<unsigned, sycl::rounding_mode::rtz>()[0];
  u = __half2uint_rz(h);
  // CHECK: ull = sycl::vec<sycl::half, 1>(h).convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __half2ull_rd(h);
  // CHECK: ull = sycl::vec<sycl::half, 1>(h).convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __half2ull_rn(h);
  // CHECK: ull = sycl::vec<sycl::half, 1>(h).convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __half2ull_ru(h);
  // CHECK: ull = sycl::vec<sycl::half, 1>(h).convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __half2ull_rz(h);
  // CHECK: us = sycl::vec<sycl::half, 1>(h).convert<unsigned short, sycl::rounding_mode::rtn>()[0];
  us = __half2ushort_rd(h);
  // CHECK: us = sycl::vec<sycl::half, 1>(h).convert<unsigned short, sycl::rounding_mode::rte>()[0];
  us = __half2ushort_rn(h);
  // CHECK: us = sycl::vec<sycl::half, 1>(h).convert<unsigned short, sycl::rounding_mode::rtp>()[0];
  us = __half2ushort_ru(h);
  // CHECK: us = sycl::vec<sycl::half, 1>(h).convert<unsigned short, sycl::rounding_mode::rtz>()[0];
  us = __half2ushort_rz(h);
  // CHECK: s = sycl::bit_cast<short, sycl::half>(h);
  s = __half_as_short(h);
  // CHECK: us = sycl::bit_cast<unsigned short, sycl::half>(h);
  us = __half_as_ushort(h);
  // CHECK: h2 = sycl::half2(h, h);
  h2 = __halves2half2(h, h);
  // CHECK: f = h2[1];
  f = __high2float(h2);
  // CHECK: h = h2[1];
  h = __high2half(h2);
  // CHECK: h2 = sycl::half2(h2[1]);
  h2 = __high2half2(h2);
  // CHECK: h2 = sycl::half2(h2[1], h2[1]);
  h2 = __highs2half2(h2, h2);
  // CHECK: h = sycl::vec<int, 1>(i).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __int2half_rd(i);
  // CHECK: h = sycl::vec<int, 1>(i).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __int2half_rn(i);
  // CHECK: h = sycl::vec<int, 1>(i).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __int2half_ru(i);
  // CHECK: h = sycl::vec<int, 1>(i).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __int2half_rz(i);
  // CHECK: h = sycl::vec<long long, 1>(ll).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __ll2half_rd(ll);
  // CHECK: h = sycl::vec<long long, 1>(ll).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __ll2half_rn(ll);
  // CHECK: h = sycl::vec<long long, 1>(ll).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __ll2half_ru(ll);
  // CHECK: h = sycl::vec<long long, 1>(ll).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __ll2half_rz(ll);
  // CHECK: f = h2[0];
  f = __low2float(h2);
  // CHECK: f = (*(&h2))[0];
  f = __low2float(*(&h2));
  // CHECK: h = h2[0];
  h = __low2half(h2);
  // CHECK: h2 = sycl::half2(h2[0]);
  h2 = __low2half2(h2);
  // CHECK: h2 = sycl::half2(h2[1], h2[0]);
  h2 = __lowhigh2highlow(h2);
  // CHECK: h2 = sycl::half2(h2[0], h2[0]);
  h2 = __lows2half2(h2, h2);
  // CHECK: h = sycl::vec<short, 1>(s).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __short2half_rd(s);
  // CHECK: h = sycl::vec<short, 1>(s).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __short2half_rn(s);
  // CHECK: h = sycl::vec<short, 1>(s).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __short2half_ru(s);
  // CHECK: h = sycl::vec<short, 1>(s).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __short2half_rz(s);
  // CHECK: h = sycl::bit_cast<sycl::half, short>(s);
  h = __short_as_half(s);
  // CHECK: h = sycl::vec<unsigned, 1>(u).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __uint2half_rd(u);
  // CHECK: h = sycl::vec<unsigned, 1>(u).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __uint2half_rn(u);
  // CHECK: h = sycl::vec<unsigned, 1>(u).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __uint2half_ru(u);
  // CHECK: h = sycl::vec<unsigned, 1>(u).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __uint2half_rz(u);
  // CHECK: h = sycl::vec<unsigned long long, 1>(ull).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __ull2half_rd(ull);
  // CHECK: h = sycl::vec<unsigned long long, 1>(ull).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __ull2half_rn(ull);
  // CHECK: h = sycl::vec<unsigned long long, 1>(ull).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __ull2half_ru(ull);
  // CHECK: h = sycl::vec<unsigned long long, 1>(ull).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __ull2half_rz(ull);
  // CHECK: h = sycl::vec<unsigned short, 1>(us).convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __ushort2half_rd(us);
  // CHECK: h = sycl::vec<unsigned short, 1>(us).convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __ushort2half_rn(us);
  // CHECK: h = sycl::vec<unsigned short, 1>(us).convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __ushort2half_ru(us);
  // CHECK: h = sycl::vec<unsigned short, 1>(us).convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __ushort2half_rz(us);
  // CHECK: h = sycl::bit_cast<sycl::half, unsigned short>(us);
  h = __ushort_as_half(us);
}

int main() { return 0; }
